#include "hip/hip_runtime.h"
// Implementation (nearly) that from Arbor.

#include <cstddef>
#include <cstdint>
#include <memory>

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
#error "Require compute capability >= 6.0"
#endif

namespace arb {
    constexpr unsigned mask_all = 0xFFFFFFFF;
    constexpr unsigned threads_per_warp = 32;

    // Return largest n = 2^k s.t. n <= i.
    // Precondition: i>0
    __device__ __inline__
    unsigned rounddown_power_of_2(std::uint32_t i) {
        return 1u<<(31u - __clz(i));
    }

    struct run_length {
        unsigned left;
        unsigned right;
        unsigned shift;
        unsigned lane_id;
        unsigned key_mask;

        __device__ __inline__
        bool is_root() const {
            return left == lane_id;
        }

        __device__
        run_length(int idx, unsigned mask) {
            key_mask = mask;
            lane_id = threadIdx.x%threads_per_warp;
            unsigned num_lanes = threads_per_warp-__clz(key_mask);

            auto right_limit = [num_lanes] (unsigned roots, unsigned shift) {
                unsigned zeros_right  = __ffs(roots>>shift);
                return zeros_right ? shift -1 + zeros_right : num_lanes;
            };

            // determine if this thread is the root (i.e. first thread with this key)
            int left_idx  = __shfl_up_sync(key_mask, idx, lane_id? 1: 0);
            int is_root = 1;
            if(lane_id>0) {
                is_root = (left_idx != idx);
            }

            // determine the range this thread contributes to
            unsigned roots = __ballot_sync(key_mask, is_root);

            // determine the bounds of the lanes with the same key as idx
            right = right_limit(roots, lane_id+1);
            left  = threads_per_warp-1-right_limit(__brev(roots), threads_per_warp-1-lane_id);

            // find the largest power of two that is less than or equal to the run length
            shift = rounddown_power_of_2(right - left);
        }
    };

    template <typename T, typename I>
    __global__
    void reduce_impl(std::size_t N, T* p, const T* v, const I* index) {
        unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;
        unsigned mask = __ballot_sync(mask_all, tid<N);

        if (tid<N) {
            auto contribution = v[tid];
            auto i = index[tid];

            run_length run(i, mask);
            unsigned shift = run.shift;
            const unsigned key_lane = run.lane_id - run.left;

            bool participate = run.shift && run.lane_id+shift<run.right;

            while (__any_sync(run.key_mask, shift)) {
                const unsigned w = participate? shift: 0;
                const unsigned source_lane = run.lane_id + w;

                T source_value = __shfl_sync(run.key_mask, contribution, source_lane);
                if (participate) {
                    contribution += source_value;
                }

                shift >>= 1;
                participate = key_lane<shift;
            }

            if (run.is_root()) {
                atomicAdd(p+i, contribution);
            }
        }
    }
} // namespace arb

template <typename T>
struct block {
    T* data = nullptr;
    std::size_t n = 0;
};

struct gpu_block_delete {
    template <typename T>
    void operator()(block<T>* bp) {
        hipFree((void*)bp->data);
        delete bp;
    }
};

template <typename T>
using gpu_block_ptr = std::unique_ptr<block<T>, gpu_block_delete>;

template <typename T>
gpu_block_ptr<T> on_gpu(block<T> b) {
    void* gpu_data = nullptr;
    hipMalloc(&gpu_data, b.n*sizeof(T));
    hipMemcpy(gpu_data,b.data, b.n*sizeof(T), hipMemcpyHostToDevice);

    auto p = gpu_block_ptr<T>(new block<T>);
    p->data = (T*)gpu_data;
    p->n = b.n;
    return p;
}

template <typename T>
void from_gpu(block<T> b, const gpu_block_ptr<T>& p) {
    if (b.n!=p->n) throw std::runtime_error("block size mismatch");
    hipMemcpy(b.data, p->data, b.n*sizeof(T), hipMemcpyDeviceToHost);
}

void arbor_cuda_reduce_impl(std::size_t N, double* p, const double* v, const int* index, int reps) {
    unsigned bwidth = 128;
    unsigned bcount = (N+bwidth-1)/bwidth;

    block<double> p_view{p, N};
    block<const double> v_view{v, N};
    block<const int> i_view{index, N};

    auto p_gpu = on_gpu(p_view);
    auto v_gpu = on_gpu(v_view);
    auto i_gpu = on_gpu(i_view);

    for (int c = 0; c<reps; ++c) {
        arb::reduce_impl<<<bcount, bwidth>>>(N, p_gpu->data, v_gpu->data, i_gpu->data);
    }

    from_gpu(p_view, p_gpu);
}


