#include "hip/hip_runtime.h"
// Implementation (nearly) that from Arbor.

#include <cstddef>
#include <cstdint>
#include <memory>

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
#error "Require compute capability >= 6.0"
#endif

namespace arb {
    constexpr unsigned mask_all = 0xFFFFFFFF;
    constexpr unsigned threads_per_warp = 32;

    // Return largest n = 2^k s.t. n <= i.
    // Precondition: i>0
    __device__ __inline__
    unsigned rounddown_power_of_2(std::uint32_t i) {
        return 1u<<(31u - __clz(i));
    }

    struct run_length {
        unsigned left;
        unsigned right;
        unsigned shift;
        unsigned lane_id;
        unsigned key_mask;

        __device__ __inline__
        bool is_root() const {
            return left == lane_id;
        }

        __device__
        run_length(int idx, unsigned mask) {
            key_mask = mask;
            lane_id = threadIdx.x%threads_per_warp;
            unsigned num_lanes = threads_per_warp-__clz(key_mask);

            auto right_limit = [num_lanes] (unsigned roots, unsigned shift) {
                unsigned zeros_right  = __ffs(roots>>shift);
                return zeros_right ? shift -1 + zeros_right : num_lanes;
            };

            // determine if this thread is the root (i.e. first thread with this key)
            int left_idx  = __shfl_up_sync(key_mask, idx, lane_id? 1: 0);
            int is_root = 1;
            if(lane_id>0) {
                is_root = (left_idx != idx);
            }

            // determine the range this thread contributes to
            unsigned roots = __ballot_sync(key_mask, is_root);

            // determine the bounds of the lanes with the same key as idx
            right = right_limit(roots, lane_id+1);
            left  = threads_per_warp-1-right_limit(__brev(roots), threads_per_warp-1-lane_id);

            // find the largest power of two that is less than or equal to the run length
            shift = rounddown_power_of_2(right - left);
        }
    };

    template <typename T, typename I>
    __global__
    void reduce_impl(std::size_t N, T* p, const T* v, const I* index) {
        unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;
        unsigned mask = __ballot_sync(mask_all, tid<N);

        if (tid<N) {
            auto contribution = v[tid];
            auto i = index[tid];

            run_length run(i, mask);
            unsigned shift = run.shift;
            const unsigned key_lane = run.lane_id - run.left;

            bool participate = run.shift && run.lane_id+shift<run.right;

            while (__any_sync(run.key_mask, shift)) {
                const unsigned w = participate? shift: 0;
                const unsigned source_lane = run.lane_id + w;

                T source_value = __shfl_sync(run.key_mask, contribution, source_lane);
                if (participate) {
                    contribution += source_value;
                }

                shift >>= 1;
                participate = key_lane<shift;
            }

            if (run.is_root()) {
                atomicAdd(p+i, contribution);
            }
        }
    }
} // namespace arb

namespace expr1 {
    constexpr unsigned mask_all = 0xFFFFFFFF;
    constexpr unsigned threads_per_warp = 32;
    constexpr unsigned log2_tpw = 5;

    template <typename T, typename I>
    __global__
    void reduce_impl(std::size_t N, T* p, const T* v, const I* index) {
        unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;

        unsigned mask = __ballot_sync(mask_all, tid<N);
        if (tid>=N) return;

	unsigned lane_id = threadIdx.x%threads_per_warp;

	I i = index[tid];
	auto i_prev = __shfl_up_sync(mask, i, 1);

	bool root = lane_id==0 || i!=i_prev;
	unsigned roots = __ballot_sync(mask, root);
	roots |= ~mask;

	unsigned width = __ffs(roots>>lane_id>>1);
	if (!width) width=threads_per_warp-lane_id;

	auto v_local = v[tid];
	auto accum = v_local;
	unsigned shift = 1;

	for (unsigned k = 0; k<log2_tpw; ++k) {
	    if (shift>=width) shift = 0;
	    T v_shift = __shfl_down_sync(mask, accum, shift);

	    if (shift) accum += v_shift;
	    shift *= 2;
	}

	if (root) atomicAdd(p+i, accum);
    }
}

namespace expr2 {
    constexpr unsigned mask_all = 0xFFFFFFFF;
    constexpr unsigned threads_per_warp = 32;
    constexpr unsigned log2_tpw = 5;

    template <typename T, typename I>
    __global__
    void reduce_impl(std::size_t N, T* p, const T* v, const I* index) {
        unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;

        unsigned mask = __ballot_sync(mask_all, tid<N);
        if (tid>=N) return;

	unsigned lane_id = threadIdx.x%threads_per_warp;

	I i = index[tid];
	auto i_prev = __shfl_up_sync(mask, i, 1);

	bool root = lane_id==0 || i!=i_prev;
	unsigned roots = __ballot_sync(mask, root);
	roots |= ~mask;

        auto v_local = v[tid];

        if (roots!=-1) {
            unsigned width = __ffs(roots>>lane_id>>1);
            if (!width) width=threads_per_warp-lane_id;

            auto accum = v_local;
            unsigned shift = 1;

            while (__any_sync(mask, shift)) {
                if (shift>=width) shift = 0;
                T v_shift = __shfl_down_sync(mask, accum, shift);

                if (shift) accum += v_shift;
                shift *= 2;
            }

            if (root) atomicAdd(p+i, accum);
        }
        else {
            atomicAdd(p+i, v_local);
        }
    }
}


template <typename T>
struct block {
    T* data = nullptr;
    std::size_t n = 0;
};

struct gpu_block_delete {
    template <typename T>
    void operator()(block<T>* bp) {
        hipFree((void*)bp->data);
        delete bp;
    }
};

template <typename T>
using gpu_block_ptr = std::unique_ptr<block<T>, gpu_block_delete>;

template <typename T>
gpu_block_ptr<T> on_gpu(block<T> b) {
    void* gpu_data = nullptr;
    hipMalloc(&gpu_data, b.n*sizeof(T));
    hipMemcpy(gpu_data, b.data, b.n*sizeof(T), hipMemcpyHostToDevice);

    auto p = gpu_block_ptr<T>(new block<T>);
    p->data = (T*)gpu_data;
    p->n = b.n;
    return p;
}

template <typename T>
void from_gpu(block<T> b, const gpu_block_ptr<T>& p) {
    if (b.n!=p->n) throw std::runtime_error("block size mismatch");
    hipMemcpy(b.data, p->data, b.n*sizeof(T), hipMemcpyDeviceToHost);
}

struct cuda_event_delete {
    void operator()(hipEvent_t* evptr) {
	if (evptr) hipEventDestroy(*evptr);
	delete evptr;
    }
};

using cuda_event = std::unique_ptr<hipEvent_t, cuda_event_delete>;
cuda_event make_cuda_event() {
    hipEvent_t* p = new hipEvent_t;
    hipEventCreate(p);
    return cuda_event(p);
}

float elapsed_time_s(const cuda_event& e0, const cuda_event& e1) {
    float t = 0;
    hipEventElapsedTime(&t, *e0, *e1);
    return t*0.001;
}

using impl_fn_ptr = void (*)(std::size_t, double*, const double*, const int*);

float impl_harness(impl_fn_ptr fn, std::size_t N, double* p, const double* v, const int* index, int reps) {
    unsigned bwidth = 128;
    unsigned bcount = (N+bwidth-1)/bwidth;

    block<double> p_view{p, N};
    block<const double> v_view{v, N};
    block<const int> i_view{index, N};

    auto p_gpu = on_gpu(p_view);
    auto v_gpu = on_gpu(v_view);
    auto i_gpu = on_gpu(i_view);

    auto e0 = make_cuda_event();
    auto e1 = make_cuda_event();

    p_gpu = on_gpu(p_view);
    hipEventRecord(*e0);
    for (int c = 0; c<reps; ++c) {
        (*fn)<<<bcount, bwidth>>>(N, p_gpu->data, v_gpu->data, i_gpu->data);
    }
    hipEventRecord(*e1);

    from_gpu(p_view, p_gpu);
    return elapsed_time_s(e0, e1);
}

float arbor_cuda_reduce_impl(std::size_t N, double* p, const double* v, const int* index, int reps) {
    return impl_harness(&arb::reduce_impl<double, int>, N, p, v, index, reps);
}

float expr1_cuda_reduce_impl(std::size_t N, double* p, const double* v, const int* index, int reps) {
    return impl_harness(&expr1::reduce_impl<double, int>, N, p, v, index, reps);
}

float expr2_cuda_reduce_impl(std::size_t N, double* p, const double* v, const int* index, int reps) {
    return impl_harness(&expr2::reduce_impl<double, int>, N, p, v, index, reps);
}
